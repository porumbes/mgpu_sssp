#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wunused-result"

#include "sssp_cpu.hxx"
#include "sssp_1gpu.hxx"
#include "sssp_mgpu.hxx"

#include <hip/hip_runtime.h>               /* for Gpuinfo */
#include <hip/hip_runtime_api.h>   /* for Gpuinfo */
#include <iomanip>
#include <cstdlib>
#include <fstream>
#include "json.hpp"
using json = nlohmann::json;

//#define RUN_CPU

// --
// Global defs

typedef int Int;
typedef float Real;

// graph
Int n_nodes;
Int n_edges;
Int* indptr;
Int* rindices;
Int* cindices;
Real* data;

void load_data(std::string inpath) {
    FILE *ptr;
    ptr = fopen(inpath.c_str(), "rb");

    fread(&n_nodes,   sizeof(Int), 1, ptr);
    fread(&n_nodes,   sizeof(Int), 1, ptr);
    fread(&n_edges,   sizeof(Int), 1, ptr);

    indptr   = (Int*)  malloc(sizeof(Int)  * (n_nodes + 1)  );
    cindices = (Int*)  malloc(sizeof(Int)  * n_edges         );
    rindices = (Int*)  malloc(sizeof(Int)  * n_edges         );
    data     = (Real*) malloc(sizeof(Real) * n_edges         );

    fread(indptr,  sizeof(Int),   n_nodes + 1 , ptr);  // send directy to the memory since thats what the thing is.
    fread(cindices, sizeof(Int),  n_edges      , ptr);
    fread(data,    sizeof(Real),  n_edges      , ptr);
    
    for(Int src = 0; src < n_nodes; src++) {
        for(Int offset = indptr[src]; offset < indptr[src + 1]; offset++) {
            rindices[offset] = src;
        }
    }
}

json gpu_info_json() {
    json j;
    hipDeviceProp_t devProps;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)   /* no valid devices */
    {
        return j;        /* empty */
    }
    int dev = 0;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&devProps, dev);
    j["gpuinfo"]["name"] = devProps.name;
    j["gpuinfo"]["total_global_mem"] = int64_t(devProps.totalGlobalMem);
    j["gpuinfo"]["major"] = devProps.major;
    j["gpuinfo"]["minor"] = devProps.minor;
    j["gpuinfo"]["clock_rate"] = devProps.clockRate;
    j["gpuinfo"]["multi_processor_count"] = devProps.multiProcessorCount;

    int runtimeVersion, driverVersion;
    hipRuntimeGetVersion(&runtimeVersion);
    hipDriverGetVersion(&driverVersion);
    j["gpuinfo"]["driver_api"] = CUDA_VERSION;
    j["gpuinfo"]["driver_version"] = driverVersion;
    j["gpuinfo"]["runtime_version"] = runtimeVersion;
    j["gpuinfo"]["compute_version"] = devProps.major * 10 + devProps.minor;

    return j;    
}

int main(int n_args, char** argument_array) {
    int n_gpus = 1;
    hipGetDeviceCount(&n_gpus);
    
    // ---------------- INPUT ----------------
    // main path_to_dataset.bin num_seeds path_to_output.json
    if(n_args < 1 || n_args != 4) {
        std::cout << "Usage: main <input_dataset.bin> <num_seeds> <path_to_output.json>\n";
	std::exit(EXIT_FAILURE);
    }

    load_data(argument_array[1]);
    int n_seeds = 1;
    n_seeds = (int)atoi(argument_array[2]);

    Int* seeds = (Int*)malloc(n_seeds * sizeof(Int));
    for(Int seed = 0; seed < n_seeds; seed++) {
        seeds[seed] = seed;
    }
    
    // ---------------- CPU ----------------
    
    Real* cpu_dist = (Real*)malloc(n_nodes * sizeof(Real));
    long long cpu_time = 0;
#ifdef RUN_CPU
    cpu_time = sssp_cpu(cpu_dist, n_seeds, seeds, n_nodes, n_edges, indptr, cindices, data);
#endif
    
    // ---------------- GPU ----------------
    
    Real* gpu_dist = (Real*)malloc(n_nodes * sizeof(Real));
    long long gpu_time = 0;
    if(n_gpus == 1) {
        gpu_time = sssp_1gpu(gpu_dist, n_seeds, seeds, n_nodes, n_edges, rindices, cindices, data);
    } else {
        gpu_time = sssp_mgpu(gpu_dist, n_seeds, seeds, n_nodes, n_edges, rindices, cindices, data, n_gpus);
    }

    for(Int i = 0; i < min(n_nodes, 40); i++) std::cout << cpu_dist[i] << " ";
    std::cout << std::endl;
    for(Int i = 0; i < min(n_nodes, 40); i++) std::cout << gpu_dist[i] << " ";
    std::cout << std::endl;

    // ---------------- VALIDATE ----------------
    
    int n_errors = 0;
#ifdef RUN_CPU
    for(Int i = 0; i < n_nodes; i++) {
        if(cpu_dist[i] != gpu_dist[i]) n_errors++;
    }
#endif
    
    std::cout << "n_seeds=" << n_seeds 
	    << " | cpu_time=" << cpu_time 
	    << " | gpu_time_microseconds=" << gpu_time 
	    << " | n_errors=" << n_errors 
	    << " | n_gpus=" << n_gpus << std::endl;
    std::cout << "dataset=" << argument_array[1] << '\n'
	      << "num-vertices=" << n_nodes << '\n'
	      << "num-edges=" << n_edges << '\n';
    
    auto j = gpu_info_json();
    j["primitive"] = "vn";
    j["graph-file"] = std::string(argument_array[1]);
    j["num_gpus"] = n_gpus;
    j["graph-edges"] = n_edges;
    j["graph-nodes"] = n_nodes;
    j["gpu-elapsed-ms"] = (double)gpu_time / 1000.0;
    time_t now = time(NULL);
    j["time"] = ctime(&now);
    j["variant"] = std::string("num_seeds:") + std::to_string(n_seeds);

    // get the dataset from the json
    auto dataset = std::string(argument_array[3]);
    std::size_t p1 = dataset.find("vn__") + 4; // skip the expected "vn__"
    std::size_t p2 = dataset.find("__GPU");
    j["dataset"] = dataset.substr(p1, p2-p1);

    //std::cout << '\n' << std::setw(4) << j << '\n';
    std::ofstream output_json(argument_array[3]);
    output_json << std::setw(4) << j << std::endl;

    return 0;
}
